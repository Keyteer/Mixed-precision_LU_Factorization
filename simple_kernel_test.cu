#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"

int main() {
    const int rows = 3, cols = 2;
    const int total_elements = rows * cols;
    
    // Allocate host memory
    fp16 *h_panel = new fp16[total_elements];
    int *h_ipiv = new int[cols];
    
    // Initialize test data
    for (int i = 0; i < total_elements; ++i) {
        h_panel[i] = double_to_fp16(i + 1.0); // Simple test values
    }
    
    // Print input
    std::cout << "Input panel:\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << fp16_to_double(h_panel[j * rows + i]) << " ";
        }
        std::cout << "\n";
    }
    
    // Allocate device memory
    fp16 *d_panel;
    int *d_ipiv;
    hipMalloc(&d_panel, total_elements * sizeof(fp16));
    hipMalloc(&d_ipiv, cols * sizeof(int));
    
    // Copy to device
    hipMemcpy(d_panel, h_panel, total_elements * sizeof(fp16), hipMemcpyHostToDevice);
    
    // Launch kernel with fewer threads for easier debugging
    int threads = std::min(32, rows - 1); // Use fewer threads
    std::cout << "Launching kernel with " << threads << " threads\n";
    
    if (threads > 0) {
        HGETF2_kernel<<<1, threads>>>(d_panel, rows, rows, cols, d_ipiv);
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
            return -1;
        }
    }
    
    // Copy results back
    hipMemcpy(h_panel, d_panel, total_elements * sizeof(fp16), hipMemcpyDeviceToHost);
    hipMemcpy(h_ipiv, d_ipiv, cols * sizeof(int), hipMemcpyDeviceToHost);
    
    // Print results
    std::cout << "Output panel:\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << fp16_to_double(h_panel[j * rows + i]) << " ";
        }
        std::cout << "\n";
    }
    
    std::cout << "IPIV: ";
    for (int i = 0; i < cols; ++i) {
        std::cout << h_ipiv[i] << " ";
    }
    std::cout << "\n";
    
    // Cleanup
    delete[] h_panel;
    delete[] h_ipiv;
    hipFree(d_panel);
    hipFree(d_ipiv);
    
    return 0;
}
