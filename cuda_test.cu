#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    std::cout << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    std::cout << "Device count: " << deviceCount << std::endl;
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Device 0: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    }
    
    return 0;
}
