#include <iostream>
#include <hip/hip_runtime.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"

int main() {
    // Test with the same 4x4 matrix from matrix_4x4.txt
    const int rows = 4, cols = 4;
    double matrix_data[] = {
        8.3, 8.6, 7.7, 1.5,  // Column 0
        9.3, 3.5, 8.6, 9.2,  // Column 1  
        4.9, 2.1, 6.2, 2.7,  // Column 2
        9.0, 5.9, 6.3, 2.6   // Column 3
    };
    
    // Print original matrix
    std::cout << "Original matrix (column-major):\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix_data[j * rows + i] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
    
    // Allocate host memory
    fp16 *h_panel = new fp16[rows * cols];
    int *h_ipiv = new int[cols];
    
    // Convert to FP16
    for (int i = 0; i < rows * cols; ++i) {
        h_panel[i] = double_to_fp16(matrix_data[i]);
    }
    
    // Allocate device memory
    fp16 *d_panel;
    int *d_ipiv;
    hipMalloc(&d_panel, rows * cols * sizeof(fp16));
    hipMalloc(&d_ipiv, cols * sizeof(int));
    
    // Copy to device
    hipMemcpy(d_panel, h_panel, rows * cols * sizeof(fp16), hipMemcpyHostToDevice);
    
    // Launch kernel with 1 block, multiple threads
    std::cout << "Launching HGETF2_kernel...\n";
    HGETF2_kernel<<<1, rows>>>(d_panel, rows, rows, cols, d_ipiv);
    
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    
    // Copy results back
    hipMemcpy(h_panel, d_panel, rows * cols * sizeof(fp16), hipMemcpyDeviceToHost);
    hipMemcpy(h_ipiv, d_ipiv, cols * sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "IPIV from HGETF2_kernel: ";
    for (int i = 0; i < cols; ++i) {
        std::cout << h_ipiv[i] << " ";
    }
    std::cout << "\n\n";
    
    std::cout << "Matrix after HGETF2_kernel:\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << fp16_to_double(h_panel[j * rows + i]) << " ";
        }
        std::cout << "\n";
    }
    
    // Cleanup
    delete[] h_panel;
    delete[] h_ipiv;
    hipFree(d_panel);
    hipFree(d_ipiv);
    
    return 0;
}
