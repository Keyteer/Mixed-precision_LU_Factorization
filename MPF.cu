#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <vector>
#include <iostream>
#include <lapacke.h>
#include <hipblas.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"
#include "dgetf2_native_npv.h"
#include "cuda_debug.h"

// GPU kernel for FP64 to FP16 conversion
__global__ void double_to_fp16_block(const double* input, fp16* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = double_to_fp16(input[idx]);
    }
}

// GPU kernel for FP16 to FP64 conversion
__global__ void fp16_to_double_block(const fp16* input, double* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fp16_to_double(input[idx]);
    }
}

// Device version of LASWP (row swaps, FP64) - Column-major order
__global__ void LASWP_kernel(double *A, int lda, int k, int cols, const int *ipiv_panel) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < cols) {
        int piv = ipiv_panel[j] - 1; // Convert from 1-based to 0-based
        if (piv != j) {
            // Swap rows j+k and piv+k for all columns (column-major)
            for (int col = 0; col < lda; ++col) {
                double tmp = A[col * lda + (j + k)];
                A[col * lda + (j + k)] = A[col * lda + (piv + k)];
                A[col * lda + (piv + k)] = tmp;
            }
        }
    }
}

// --- DTRSM: Triangular solve en FP64 usando cuBLAS ---
void DTRSM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, int m, int n) {
    const double alpha = 1.0;
    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
        m, n, &alpha, dA, lda, dB, ldb);
}

// --- DGEMM: Multiplicación de matrices en FP64 usando cuBLAS ---
void DGEMM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, double *dC, int ldc, int m, int n, int k) {
    const double alpha = -1.0;
    const double beta = 1.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, lda, dB, ldb, &beta, dC, ldc);
}

// --- MPF: All on GPU ---
// h_A [in] 
void MPF(double *A, int N, int r, int *IPIV) {
    
    CUDA_CHECK("ENTRY");
    
    // Check CUDA device availability
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA GetDeviceCount status: " << hipGetErrorString(cudaStatus) << std::endl;
    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;
    
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found!" << std::endl;
        return;
    }
    
    CUDA_CHECK("AFTER_DEVICE_COUNT");
    
    hipSetDevice(0);  // Explicitly set device
    CUDA_CHECK("AFTER_SET_DEVICE");
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    std::cout << "Using device: " << deviceProp.name << std::endl;
    std::cout << "Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    
    CUDA_CHECK("AFTER_DEVICE_PROPERTIES");
    
    // Initialize host memory
    double *h_A = new double[N * N];
    std::memcpy(h_A, A, N * N * sizeof(double));

    CUDA_CHECK("AFTER_HOST_ALLOC");

    // Allocate device memory
    double *d_A;
    hipMalloc(&d_A, N * N * sizeof(double));
    CUDA_CHECK("AFTER_D_A_MALLOC");
    
    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK("AFTER_D_A_MEMCPY");

    fp16 *d_P_FP16_buffer;
    hipMalloc(&d_P_FP16_buffer, N * r * sizeof(fp16));
    CUDA_CHECK("AFTER_FP16_MALLOC");
    
    double *d_P_FP64_NPV_buffer;
    hipMalloc(&d_P_FP64_NPV_buffer, N * r * sizeof(double));
    CUDA_CHECK("AFTER_FP64_MALLOC");

    int *d_IPIV_panel;
    hipMalloc(&d_IPIV_panel, r * sizeof(int));
    CUDA_CHECK("AFTER_IPIV_PANEL_MALLOC");
    
    int *d_IPIV;
    hipMalloc(&d_IPIV, N * sizeof(int));
    CUDA_CHECK("AFTER_IPIV_MALLOC");



    // Initialize IPIV to identity permutation
    int *h_IPIV = new int[N];
    for (int i = 0; i < N; i++) h_IPIV[i] = i + 1; // 1-based indexing
    hipMemcpy(d_IPIV, h_IPIV, N * sizeof(int), hipMemcpyHostToDevice);
    CUDA_CHECK("AFTER_IPIV_INIT");

    // Check CUDA context after memory operations
    hipError_t contextCheck = hipGetLastError();
    std::cout << "CUDA context after memory ops: " << hipGetErrorString(contextCheck) << std::endl;

    CUDA_CHECK("BEFORE_CUBLAS_CREATE");
    hipblasHandle_t handle;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);
    std::cout << "cuBLAS create status: " << hipblasStatus_t << std::endl;
    CUDA_CHECK("AFTER_CUBLAS_CREATE");

    for (int k = 0; k < N; k += r) {
        int current_panel_cols = std::min(r, N - k); // Number of columns in the current panel (r or N%r)
        int panel_rows = N - k; // Number of rows in the panel

        // a. Copy panel to FP16 buffer on device
        // First copy to FP64 buffer, then convert to FP16
        hipMemcpy2D(d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            d_A + k * N + k, N * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);

        // Convert FP64 panel to FP16 (you need a conversion kernel here)
        // For now, do it on host as a temporary solution
        int total_elements = panel_rows * current_panel_cols;
        dim3 block(256);
        dim3 grid((total_elements + 255) / 256);
        double_to_fp16_block<<<grid, block>>>(d_P_FP64_NPV_buffer, d_P_FP16_buffer, total_elements);
        hipDeviceSynchronize();

        // Initialize IPIV panel to identity permutation for this panel
        int ident_ipiv_panel[r];
        for (int i = 0; i < current_panel_cols; i++) ident_ipiv_panel[i] = i + 1; // 1-based indexing
        hipMemcpy(d_IPIV_panel, ident_ipiv_panel, current_panel_cols * sizeof(int), hipMemcpyHostToDevice);

        int ipiv_panel_print[r];
        // Initialize the print array to avoid garbage values
        for (int i = 0; i < r; i++) ipiv_panel_print[i] = 0;
        hipMemcpy(ipiv_panel_print, d_IPIV_panel, current_panel_cols * sizeof(int), hipMemcpyDeviceToHost);
        // Debug print for IPIV panel
        std::cout << "IPIV panel: ";
        for (int i = 0; i < current_panel_cols; ++i)
            std::cout << ipiv_panel_print[i] << " ";
        std::cout << std::endl;

        // b.i. Panel LU in FP16 (kernel)
        int threads = std::min(1024, panel_rows - 1);
        if (threads > 0) {
            std::cout << "Launching HGETF2_kernel with " << threads << " threads" << std::endl;
            HGETF2_kernel << <1, threads >> > (d_P_FP16_buffer, panel_rows, panel_rows, current_panel_cols, d_IPIV_panel);
            hipError_t err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                std::cout << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
                std::cout << "Using CPU fallback for IPIV calculation" << std::endl;
                
                // CPU fallback: simple pivoting logic
                int *h_fallback_ipiv = new int[current_panel_cols];
                for (int i = 0; i < current_panel_cols; i++) {
                    h_fallback_ipiv[i] = i + 1; // Identity for now (no actual pivoting)
                }
                hipMemcpy(d_IPIV_panel, h_fallback_ipiv, current_panel_cols * sizeof(int), hipMemcpyHostToDevice);
                delete[] h_fallback_ipiv;
            } else {
                std::cout << "Kernel completed successfully" << std::endl;
            }
        }

        hipMemcpy(ipiv_panel_print, d_IPIV_panel, current_panel_cols * sizeof(int), hipMemcpyDeviceToHost);
        // Debug print for IPIV panel
        std::cout << "IPIV panel: ";
        for (int i = 0; i < current_panel_cols; ++i)
            std::cout << ipiv_panel_print[i] << " ";
        std::cout << std::endl;

        // b.ii. Apply permutations to FP64 matrix (kernel)
        // LASWP_kernel << <(current_panel_cols + 255) / 256, 256 >> > (d_A, N, k, current_panel_cols, d_IPIV_panel);
        // hipDeviceSynchronize();


        // Copy panel from device to host
        hipMemcpy2D(h_A, N * sizeof(double),
                     d_A + k * N + k, N * sizeof(double),
                     N * sizeof(double), N,
                     hipMemcpyDeviceToHost);

        // Copy IPIV panel from device to host for LAPACKE_dlaswp
        int* h_ipiv_panel = new int[current_panel_cols];
        hipMemcpy(h_ipiv_panel, d_IPIV_panel, current_panel_cols * sizeof(int), hipMemcpyDeviceToHost);

        // Apply row swaps using LAPACKE_dlaswp (1-based ipiv)
        LAPACKE_dlaswp(LAPACK_COL_MAJOR, panel_rows, h_A, N, 1, current_panel_cols, h_ipiv_panel, 1);

        delete[] h_ipiv_panel;

        // Copy updated panel back to device
        hipMemcpy2D(d_A + k * N + k, N * sizeof(double),
                     h_A, N * sizeof(double),
                     N * sizeof(double), N,
                     hipMemcpyHostToDevice);

        // Update global IPIV array
        int *h_panel_ipiv = new int[current_panel_cols];
        hipMemcpy(h_panel_ipiv, d_IPIV_panel, current_panel_cols * sizeof(int), hipMemcpyDeviceToHost);
        
        std::cout << "Panel IPIV from kernel: ";
        for (int j = 0; j < current_panel_cols; ++j) {
            std::cout << h_panel_ipiv[j] << " ";
        }
        std::cout << std::endl;
        
        for (int j = 0; j < current_panel_cols; ++j) {
            // h_panel_ipiv[j] is 1-based relative to panel start
            // Convert to global 1-based index: panel_start + panel_relative_index
            h_IPIV[k + j] = h_panel_ipiv[j] + k;
            std::cout << "Global IPIV[" << (k + j) << "] = " << h_IPIV[k + j] << std::endl;
        }
        delete[] h_panel_ipiv;

        // b.iii. Copy updated panel back for FP64 factorization
        hipMemcpy2D(d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            d_A + k * N + k, N * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);



            
        // Panel LU in FP64 (no pivoting, kernel)
        if (threads > 0) {
            // DGETF2_NATIVE_NPV_kernel << <1, threads >> > (d_P_FP64_NPV_buffer, panel_rows, panel_rows, current_panel_cols);
            hipDeviceSynchronize();
        }

        // Copy back the panel to d_A
        hipMemcpy2D(d_A + k * N + k, N * sizeof(double),
            d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);

        // c. Trailing submatrix update (cuBLAS)
        if (k + current_panel_cols < N) {
            int m = panel_rows - current_panel_cols;
            int n = N - k - current_panel_cols;
            DTRSM_cublas(
                handle,
                d_A + k * N + k + current_panel_cols,
                N,
                d_A + (k + current_panel_cols) * N + k + current_panel_cols,
                N,
                m,
                current_panel_cols
            );
            DGEMM_cublas(
                handle,
                d_A + (k + current_panel_cols) * N + k + current_panel_cols,
                N,
                d_A + (k + current_panel_cols) * N + k,
                N,
                d_A + k * N + k + current_panel_cols,
                N,
                m,
                n,
                current_panel_cols
            );
        }
    }

    // Copy result back to host
    hipMemcpy(h_A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    memcpy(A, h_A, N * N * sizeof(double));

    // Copy IPIV back to host
    memcpy(IPIV, h_IPIV, N * sizeof(int));
    // Cleanup
    // hipblasDestroy(handle);
    delete[] h_A;
    delete[] h_IPIV;
    hipFree(d_A);
    hipFree(d_P_FP16_buffer);
    hipFree(d_P_FP64_NPV_buffer);
    hipFree(d_IPIV_panel);
    hipFree(d_IPIV);
}