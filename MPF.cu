#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <vector>
#include <iostream>
#include <lapacke.h>
#include <hipblas.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"
#include "dgetf2_native_npv.h"
#include "cuda_debug.h"

#define __threads_per_block__ 256

// Quick calculation of blocks needed based on the number of threads needed
int inline grid_size(int threads_needed) {
    return (threads_needed + __threads_per_block__ - 1) / __threads_per_block__;
}

// GPU kernel for FP64 to FP16 conversion
__global__ void double_to_fp16_block(const double *input, fp16 *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = double_to_fp16(input[idx]);
    }
}

// GPU kernel for FP16 to FP64 conversion
__global__ void fp16_to_double_block(const fp16 *input, double *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fp16_to_double(input[idx]);
    }
}

// Efficient device version of LASWP (row swaps, FP64) - Column-major order
// Apply swaps sequentially for each panel column
// A [in/out] pointer to the matrix A
// lda [in] leading dimension of A
// k [in] starting row index for the panel
// cols [in] number of columns in the panel
// ipiv_panel [in] array of pivot indices for the panel (1-based global indexing)
__global__ void LASWP_kernel(double *A, int lda, int k, int cols, const int *ipiv_panel) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    // Print matrix A (for debugging)
    if (col == 0 && blockIdx.y == 0 && threadIdx.y == 0) {
        // Debug: Matrix A Before Swap
        /*printf("Matrix A Before Swap:\n");
        for (int i = 0; i < lda; ++i) {
            for (int j = 0; j < lda; ++j) {
                printf("%f ", A[j * lda + i]);
            }
            printf("\n");
        }*/
    }
    if (col < lda) {
        // Apply swaps sequentially for this column
        for (int panel_col = 0; panel_col < cols; ++panel_col) {
            int current_row = k + panel_col;              // Current row being processed
            int pivot_row = ipiv_panel[panel_col] - 1;    // Convert to 0-based global index

            if (pivot_row != current_row) {
                // Swap A[col * lda + current_row] <-> A[col * lda + pivot_row]
                double tmp = A[col * lda + current_row];
                A[col * lda + current_row] = A[col * lda + pivot_row];
                A[col * lda + pivot_row] = tmp;
            }
        }
    }

    if (col == 0 && blockIdx.y == 0 && threadIdx.y == 0) {
        // Debug: Matrix A After Swap
        /*printf("Matrix A After Swap:\n");
        for (int i = 0; i < lda; ++i) {
            for (int j = 0; j < lda; ++j) {
                printf("%f ", A[j * lda + i]);
            }
            printf("\n");
        }*/
    }
}

// --- DTRSM: Triangular solve en FP64 usando cuBLAS ---
void DTRSM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, int m, int n) {
    const double alpha = 1.0;
    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
        m, n, &alpha, dA, lda, dB, ldb);
}

// --- DGEMM: Multiplicación de matrices en FP64 usando cuBLAS ---
void DGEMM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, double *dC, int ldc, int m, int n, int k) {
    const double alpha = -1.0;
    const double beta = 1.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, lda, dB, ldb, &beta, dC, ldc);
}

// --- MPF: Mixed‑precision Pre‑pivoting Factorization ---
// A [in/out] pointer to the matrix A
// N [in] size of the matrix A (N x N)
// r [in] panel size for mixed-precision factorization
// IPIV [out] array to store pivot indices (1-based global indexing)
void MPF(double *A, int N, int r, int *IPIV) {

    CUDA_CHECK("ENTRY");
    
    // Check CUDA device availability
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices available." << std::endl;
        return;
    }

    hipSetDevice(0);  // Explicitly set device


    // Allocate device memory
    double *d_A;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);

    fp16 *d_P_FP16_buffer;
    hipMalloc(&d_P_FP16_buffer, N * r * sizeof(fp16));

    double *d_P_FP64_NPV_buffer;
    hipMalloc(&d_P_FP64_NPV_buffer, N * r * sizeof(double));

    int *d_IPIV_panel;
    hipMalloc(&d_IPIV_panel, r * sizeof(int));

    int *d_IPIV;
    hipMalloc(&d_IPIV, N * sizeof(int));

    hipblasHandle_t handle;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);

    // Debug: Print d_A before panel iteration
    /*hipDeviceSynchronize();
    std::vector<double> h_A(N * N);
    hipMemcpy(h_A.data(), d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "d_A before panel iteration:" << std::endl;
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            std::cout << h_A[row + col * N] << " ";
        }
        std::cout << std::endl;
    }*/
    // Panel iteration
    for (int k = 0; k < N; k += r) {
        int panel_cols = std::min(r, N - k); // Number of columns in the current panel (r or N%r)
        int panel_rows = N - k; // Number of rows in the panel

        if (panel_rows > 1) {

            // 1.1 Extract panel from matrix A to FP64 buffer
            // Copy panel column by column using hipMemcpy
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    d_A + (k + col) * N + k,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }
            hipDeviceSynchronize();
            std::vector<double> h_P_FP64_NPV_buffer(panel_rows * panel_cols);
            hipMemcpy(h_P_FP64_NPV_buffer.data(), d_P_FP64_NPV_buffer, panel_rows * panel_cols * sizeof(double), hipMemcpyDeviceToHost);
            // Debug: d_P_FP64_NPV_buffer
            
            /*std::cout << "d_P_FP64_NPV_buffer:" << std::endl;
            for (int row = 0; row < panel_rows; ++row) {
                for (int col = 0; col < panel_cols; ++col) {
                    std::cout << h_P_FP64_NPV_buffer[row + col * panel_rows] << " ";
                }
                std::cout << std::endl;
            }*/
            // 1.2 Convert and copy FP64 panel to FP16 panel
            int total_elements = panel_rows * panel_cols;
            double_to_fp16_block << <grid_size(total_elements), __threads_per_block__ >> > (d_P_FP64_NPV_buffer, d_P_FP16_buffer, total_elements);
            hipDeviceSynchronize();



            // 2 Panel LU factorization in FP16 using Cooperative Groups
            int num_blocks = grid_size(panel_rows);
            int threads_per_block = __threads_per_block__;
            
            void* args[] = {&d_P_FP16_buffer, &panel_rows, &panel_rows, &panel_cols, &d_IPIV_panel};
            
            //Debug: print h_P_FP16_buffer
            /*
            hipDeviceSynchronize();
            std::vector<fp16> h_P_FP16_buffer(panel_rows * panel_cols);
            hipMemcpy(h_P_FP16_buffer.data(), d_P_FP16_buffer, panel_rows * panel_cols * sizeof(fp16), hipMemcpyDeviceToHost);
            std::cout << "d_P_FP16_buffer:" << std::endl;
            for (int row = 0; row < panel_rows; ++row) {
                for (int col = 0; col < panel_cols; ++col) {
                    std::cout << static_cast<float>(fp16_to_double(h_P_FP16_buffer[row + col * panel_rows])) << " ";
                }
                std::cout << std::endl;
            }*/
            
            hipError_t err = hipLaunchCooperativeKernel((void*)HGETF2_kernel, 
                                                        dim3(num_blocks), dim3(threads_per_block), 
                                                        args, 0, 0);
            if (err != hipSuccess) {
                std::cout << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
            } else {
                std::cout << "Kernel completed successfully with " << num_blocks << " blocks" << std::endl;
            }
            
            hipDeviceSynchronize();



            // 3.1 Apply permutations to FP64 matrix (kernel)
            LASWP_kernel << <grid_size(N), __threads_per_block__ >> > (d_A, N, k, panel_cols, d_IPIV_panel);
            hipDeviceSynchronize();

            // 3.2 Update global IPIV array
            int *h_panel_ipiv = new int[panel_cols];
            hipMemcpy(h_panel_ipiv, d_IPIV_panel, panel_cols * sizeof(int), hipMemcpyDeviceToHost);

            for (int j = 0; j < panel_cols; ++j) {
                // h_panel_ipiv[j] is already a global 1-based index from HGETF2_kernel
                // No need to add k again
                IPIV[k + j] = h_panel_ipiv[j] + k;
            }
            delete[] h_panel_ipiv;


            // 4.1 Copy updated panel back for FP64 factorization
            // Copy updated panel from d_A back to d_P_FP64_NPV_buffer column by column
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    d_A + (k + col) * N + k,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }

            // 4.2 Panel LU factorization in FP64 withot pivoting (kernel)
            dgetf2_native_npv << <grid_size(panel_rows), __threads_per_block__ >> > (panel_rows, panel_cols, d_P_FP64_NPV_buffer, panel_rows);
            hipDeviceSynchronize();

            // 4.3 Copy back the panel to matrix A
            // Copy back the panel to matrix A column by column
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_A + (k + col) * N + k,
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }
            // Debug: Print d_A before trailing
            /*
            hipDeviceSynchronize();
            std::vector<double> h_A(N * N);
            hipMemcpy(h_A.data(), d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
            std::cout << "d_A trailing update 1 (k = " << k << "):" << std::endl;
            for (int row = 0; row < N; ++row) {
                for (int col = 0; col < N; ++col) {
                    std::cout << h_A[row + col * N] << " ";
                }
                std::cout << std::endl;
            }*/

            // 5 Trailing submatrix update (cuBLAS)
            if (k + panel_cols < N) {
                int m = panel_rows - panel_cols;
                int n = N - k - panel_cols;
                // 5.1 Solve triangular system (DTRSM) U = L^{T} x A_trailing
                DTRSM_cublas(
                    handle,
                    d_A + k * N + k + panel_cols,
                    N,
                    d_A + (k + panel_cols) * N + k + panel_cols,
                    N,
                    m,
                    panel_cols
                );
                
                // Debug: Print d_A after panel iteration but before DGEMM
                /*hipDeviceSynchronize();
                std::vector<double> h_A(N * N);
                hipMemcpy(h_A.data(), d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
                std::cout << "d_A in middle of trailing update (k = " << k << "):" << std::endl;
                for (int row = 0; row < N; ++row) {
                    for (int col = 0; col < N; ++col) {
                        std::cout << h_A[row + col * N] << " ";
                    }
                    std::cout << std::endl;
                }            */
                // 5.2 Update trailing submatrix (DGEMM)  A_trailing = A_trailing - L x U
                DGEMM_cublas(
                    handle,
                    d_A + k * N + k + panel_cols,
                    N,
                    d_A + (k + panel_cols) * N + k,
                    N,
                    d_A + (k + panel_cols) * N + k + panel_cols,
                    N,
                    m,
                    n,
                    panel_cols
                );
                // Debug: Print d_A after trailing update
                
                hipDeviceSynchronize();
                std::vector<double> h1_A(N * N);
                hipMemcpy(h1_A.data(), d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
                std::cout << "d_A after trailing update (k = " << k << "):" << std::endl;
                for (int row = 0; row < N; ++row) {
                    for (int col = 0; col < N; ++col) {
                        std::cout << h1_A[row + col * N] << " ";
                    }
                    std::cout << std::endl;
                }    
            }
        }
        /*
        // Debug: Print d_A after panel iteration
        hipDeviceSynchronize();
        std::vector<double> h_A(N * N);
        hipMemcpy(h_A.data(), d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "d_A after panel iteration (k = " << k << "):" << std::endl;
        for (int row = 0; row < N; ++row) {
            for (int col = 0; col < N; ++col) {
                std::cout << h_A[row + col * N] << " ";
            }
            std::cout << std::endl;
        }*/
    }

    // Copy matrix back to host
    hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // Note: IPIV is already updated on host side during panel processing

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_P_FP16_buffer);
    hipFree(d_P_FP64_NPV_buffer);
    hipFree(d_IPIV_panel);
    hipFree(d_IPIV);
}