#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <vector>
#include <iostream>
#include <hip/hip_fp16.h> // Para __half y funciones de conversión FP16
#include <hip/hip_runtime.h>
#include <lapacke.h>
#include <hipblas.h> // Para cuBLAS


// Simulación de FP16 (usa __half como tipo real de CUDA)
using fp16 = __half;

__device__ inline void swap_fp16(fp16 &a, fp16 &b) {
    fp16 tmp = a;
    a = b;
    b = tmp;
}

// Conversión robusta de FP64 a FP16 con control de overflow/underflow
fp16 double_to_fp16(double x) {
    // 1. Conversion to single precision
    float xf = static_cast<float>(x);

    // 2. Overflow/underflow thresholds for FP16
    constexpr float FP16_MAX = 65504.0f;
    constexpr float FP16_MIN_POS = 6.10352e-05f; // Smallest positive normal FP16

    // 3. Overflow handling
    if (xf > FP16_MAX)
        xf = FP16_MAX;
    else if (xf < -FP16_MAX)
        xf = -FP16_MAX;

    // 4. Underflow handling
    if (xf > -FP16_MIN_POS && xf < FP16_MIN_POS)
        xf = 0.0f;

    // 5. Conversion to half precision with rounding to nearest even
    return __float2half_rn(xf);
}

// Conversión simple FP16 -> FP64 (placeholder)
double fp16_to_double(fp16 x) {
    // Aquí deberías usar una conversión real a FP64
    return static_cast<double>(static_cast<float>(x));
}

// HGETF2: Factorización LU con pivoteo parcial en FP16 (simulada)
/*void HGETF2(fp16 *panel, int ld, int rows, int cols, int *ipiv_panel) {
    for (int j = 0; j < cols; ++j) {

        // Búsqueda de pivote (máximo valor absoluto en la columna j)
        int piv = j;
        fp16 maxval = __habs(panel[j * ld + j]);
        for (int i = j + 1; i < rows; ++i) {
            fp16 val = __habs((panel[j * ld + i]));
            if (val > maxval) {
                maxval = val;
                piv = i;
            }
        }
        ipiv_panel[j] = piv;

        // Intercambio de filas si es necesario
        if (piv != j) {
            for (int k = 0; k < cols; ++k)
                swap_fp16(panel[k * ld + j], panel[k * ld + piv]);
        }

        // Factorización
        for (int i = j + 1; i < rows; ++i) {
            fp16 lij = panel[j * ld + i] / panel[j * ld + j];
            panel[j * ld + i] = lij;
            for (int k = j + 1; k < cols; ++k) {
                fp16 a = panel[k * ld + i];
                fp16 b = panel[k * ld + j];
                panel[k * ld + i] = a - b * lij;
            }
        }
    }
}*/

// Kernel de CUDA para HGETF2
__global__ void HGETF2_kernel(fp16 *panel, int ld, int rows, int cols, int *ipiv_panel) {
    int tid = threadIdx.x;
    for (int j = 0; j < cols; ++j) {
        // 1. Búsqueda de pivote (serial, por el thread 0)
        int piv = j;
        fp16 maxval = __habs(panel[j * ld + j]);
        if (tid == 0) {
            for (int i = j + 1; i < rows; ++i) {
                fp16 val = __habs(panel[j * ld + i]);
                if (val > maxval) {
                    maxval = val;
                    piv = i;
                }
            }
            ipiv_panel[j] = piv;
        }
        __syncthreads();

        // 2. Intercambio de filas (serial, por el thread 0)
        if (tid == 0 && piv != j) {
            for (int k = 0; k < cols; ++k)
                swap_fp16(panel[k * ld + j], panel[k * ld + piv]);
        }
        __syncthreads();

        // 3. Eliminación (paralelo: cada thread maneja una fila)
        int i = j + 1 + tid;
        if (i < rows) {
            fp16 lij = panel[j * ld + i] / panel[j * ld + j];
            panel[j * ld + i] = lij;
            for (int k = j + 1; k < cols; ++k) {
                fp16 a = panel[k * ld + i];
                fp16 b = panel[k * ld + j];
                panel[k * ld + i] = a - b * lij;
            }
        }
        __syncthreads();
    }
}

// DGETF2_NATIVE_NPV: Factorización LU sin pivoteo en FP64
void DGETF2_NATIVE_NPV(double* panel, int ld, int rows, int cols) {
    for (int j = 0; j < cols; ++j) {
        for (int i = j + 1; i < rows; ++i) {
            panel[j * ld + i] /= panel[j * ld + j];
            for (int k = j + 1; k < cols; ++k)
                panel[k * ld + i] -= panel[k * ld + j] * panel[j * ld + i];
        }
    }
}

// --- LASWP: Intercambio de filas en FP64 usando LAPACK ---
void LASWP(double* A, int n, int k, int cols, const int* ipiv_panel) {
    // Usar LAPACKE_dlaswp para mayor robustez
    LAPACKE_dlaswp(LAPACK_COL_MAJOR, n, A, n, k + 1, k + cols, ipiv_panel, 1);
}

// --- DTRSM: Triangular solve en FP64 usando cuBLAS ---
void DTRSM_cublas(hipblasHandle_t handle, double* dA, int lda, double* dB, int ldb, int m, int n) {
    const double alpha = 1.0;
    // Lado izquierdo, L, no transpuesta, no unitaria
    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                m, n, &alpha, dA, lda, dB, ldb);
}

// --- DGEMM: Multiplicación de matrices en FP64 usando cuBLAS ---
void DGEMM_cublas(hipblasHandle_t handle, double* dA, int lda, double* dB, int ldb, double* dC, int ldc, int m, int n, int k) {
    const double alpha = -1.0;
    const double beta = 1.0;
    // C = beta*C + alpha*A*B
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, lda, dB, ldb, &beta, dC, ldc);
}

// Algoritmo MPF
void MPF(double* A, int N, int r, std::vector<int>& IPIV) {
    // Buffers auxiliares
    std::vector<fp16> P_FP16_buffer(N * r);
    std::vector<double> P_FP64_NPV_buffer(N * r);
    std::vector<int> IPIV_panel(r);
    IPIV.resize(N);

    // --- Create cuBLAS handle ---
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int k = 0; k < N; k += r) {
        int current_panel_cols = std::min(r, N - k);
        int panel_rows = N - k;

        // a. Copiar panel a FP16
        for (int j = 0; j < current_panel_cols; ++j)
            for (int i = 0; i < panel_rows; ++i)
                P_FP16_buffer[j * panel_rows + i] = double_to_fp16(A[(k + j) * N + (k + i)]);

        // b.i. Factorización LU con pivoteo parcial en FP16
        HGETF2_kernel(P_FP16_buffer.data(), panel_rows, panel_rows, current_panel_cols, IPIV_panel.data());

        // b.ii. Aplicar permutaciones a la matriz original FP64
        for (int j = 0; j < current_panel_cols; ++j)
            IPIV[k + j] = k + IPIV_panel[j];
        LASWP(A, N, k, current_panel_cols, IPIV_panel.data());

        // b.iii. Factorización de panel sin pivoteo en FP64
        for (int j = 0; j < current_panel_cols; ++j)
            for (int i = 0; i < panel_rows; ++i)
                P_FP64_NPV_buffer[j * panel_rows + i] = A[(k + j) * N + (k + i)];
        DGETF2_NATIVE_NPV(P_FP64_NPV_buffer.data(), panel_rows, panel_rows, current_panel_cols);
        // Copiar de vuelta el panel factorizado
        for (int j = 0; j < current_panel_cols; ++j)
            for (int i = 0; i < panel_rows; ++i)
                A[(k + j) * N + (k + i)] = P_FP64_NPV_buffer[j * panel_rows + i];

        // c. Actualización de trailing submatrix usando cuBLAS
        if (k + current_panel_cols < N) {
            int m = panel_rows - current_panel_cols;
            int n = N - k - current_panel_cols;
            // DTRSM: resolver parte inferior del panel
            DTRSM_cublas(
                handle,
                &A[k * N + k + current_panel_cols], // dA (L)
                N,
                &A[(k + current_panel_cols) * N + k + current_panel_cols], // dB (parte inferior del panel)
                N,
                m,
                current_panel_cols
            );
            // DGEMM: actualizar trailing submatrix
            DGEMM_cublas(
                handle,
                &A[(k + current_panel_cols) * N + k + current_panel_cols], // dA
                N,
                &A[(k + current_panel_cols) * N + k], // dB
                N,
                &A[k * N + k + current_panel_cols], // dC
                N,
                m,
                n,
                current_panel_cols
            );
        }
    }

    // --- Destroy cuBLAS handle ---
    hipblasDestroy(handle);
}