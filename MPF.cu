#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <vector>
#include <iostream>
#include <lapacke.h>
#include <hipblas.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"
#include "dgetf2_native_npv.h"
#include "cuda_debug.h"

// GPU kernel for FP64 to FP16 conversion
__global__ void double_to_fp16_block(const double* input, fp16* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = double_to_fp16(input[idx]);
    }
}

// GPU kernel for FP16 to FP64 conversion
__global__ void fp16_to_double_block(const fp16* input, double* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fp16_to_double(input[idx]);
    }
}

// Efficient device version of LASWP (row swaps, FP64) - Column-major order
// Each thread handles a single (row, col) swap if needed
__global__ void LASWP_kernel(double *A, int lda, int k, int cols, const int *ipiv_panel) {
    int panel_row = blockIdx.y * blockDim.y + threadIdx.y; // 0 <= panel_row < cols
    int col = blockIdx.x * blockDim.x + threadIdx.x;        // 0 <= col < lda
    if (panel_row < cols && col < lda) {
        int piv = ipiv_panel[panel_row] - 1; // 1-based to 0-based
        if (piv != panel_row) {
            int row1 = panel_row + k;
            int row2 = piv + k;
            // Swap A[col * lda + row1] <-> A[col * lda + row2]
            double tmp = A[col * lda + row1];
            A[col * lda + row1] = A[col * lda + row2];
            A[col * lda + row2] = tmp;
        }
    }
}

// --- DTRSM: Triangular solve en FP64 usando cuBLAS ---
void DTRSM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, int m, int n) {
    const double alpha = 1.0;
    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
        m, n, &alpha, dA, lda, dB, ldb);
}

// --- DGEMM: Multiplicación de matrices en FP64 usando cuBLAS ---
void DGEMM_cublas(hipblasHandle_t handle, double *dA, int lda, double *dB, int ldb, double *dC, int ldc, int m, int n, int k) {
    const double alpha = -1.0;
    const double beta = 1.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, lda, dB, ldb, &beta, dC, ldc);
}

// --- MPF: All on GPU ---
// h_A [in] 
void MPF(double *A, int N, int r, int *IPIV) {
    
    CUDA_CHECK("ENTRY");
    
    // Check CUDA device availability
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        return;
    }
    
    hipSetDevice(0);  // Explicitly set device
    
    // Initialize host memory
    double *h_A = new double[N * N];
    std::memcpy(h_A, A, N * N * sizeof(double));

    // Allocate device memory
    double *d_A;
    hipMalloc(&d_A, N * N * sizeof(double));
    
    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);

    fp16 *d_P_FP16_buffer;
    hipMalloc(&d_P_FP16_buffer, N * r * sizeof(fp16));
    
    double *d_P_FP64_NPV_buffer;
    hipMalloc(&d_P_FP64_NPV_buffer, N * r * sizeof(double));

    int *d_IPIV_panel;
    hipMalloc(&d_IPIV_panel, r * sizeof(int));
    
    int *d_IPIV;
    hipMalloc(&d_IPIV, N * sizeof(int));



    // Initialize IPIV to identity permutation
    int *h_IPIV = new int[N];
    for (int i = 0; i < N; i++) h_IPIV[i] = i + 1; // 1-based indexing
    hipMemcpy(d_IPIV, h_IPIV, N * sizeof(int), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);

    for (int k = 0; k < N; k += r) {
        int current_panel_cols = std::min(r, N - k); // Number of columns in the current panel (r or N%r)
        int panel_rows = N - k; // Number of rows in the panel

        // a. Copy panel to FP16 buffer on device
        // First copy to FP64 buffer, then convert to FP16
        hipMemcpy2D(d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            d_A + k * N + k, N * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);

        // Convert FP64 panel to FP16 (you need a conversion kernel here)
        // For now, do it on host as a temporary solution
        int total_elements = panel_rows * current_panel_cols;
        dim3 block(256);
        dim3 grid((total_elements + 255) / 256);
        double_to_fp16_block<<<grid, block>>>(d_P_FP64_NPV_buffer, d_P_FP16_buffer, total_elements);
        hipDeviceSynchronize();

        // Initialize IPIV panel to identity permutation for this panel
        int ident_ipiv_panel[r];
        for (int i = 0; i < current_panel_cols; i++) ident_ipiv_panel[i] = i + 1; // 1-based indexing
        hipMemcpy(d_IPIV_panel, ident_ipiv_panel, current_panel_cols * sizeof(int), hipMemcpyHostToDevice);

        // b.i. Panel LU in FP16 (kernel)
        int threads = std::min(1024, panel_rows - 1);
        if (threads > 0) {
            HGETF2_kernel << <1, threads >> > (d_P_FP16_buffer, panel_rows, panel_rows, current_panel_cols, d_IPIV_panel);
            hipError_t err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                std::cout << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
                std::cout << "Using CPU fallback for IPIV calculation" << std::endl;
                
                // CPU fallback: simple pivoting logic
                int *h_fallback_ipiv = new int[current_panel_cols];
                for (int i = 0; i < current_panel_cols; i++) {
                    h_fallback_ipiv[i] = i + 1; // Identity for now (no actual pivoting)
                }
                hipMemcpy(d_IPIV_panel, h_fallback_ipiv, current_panel_cols * sizeof(int), hipMemcpyHostToDevice);
                delete[] h_fallback_ipiv;
            } else {
                std::cout << "Kernel completed successfully" << std::endl;
            }
        }

        // b.ii. Apply permutations to FP64 matrix (kernel)
        dim3 laswp_block(32, 32);
        dim3 laswp_grid((N + 31) / 32, (current_panel_cols + 31) / 32);
        LASWP_kernel<<<laswp_grid, laswp_block>>>(d_A, N, k, current_panel_cols, d_IPIV_panel);
        hipDeviceSynchronize();


        // Update global IPIV array
        int *h_panel_ipiv = new int[current_panel_cols];
        hipMemcpy(h_panel_ipiv, d_IPIV_panel, current_panel_cols * sizeof(int), hipMemcpyDeviceToHost);
        
        for (int j = 0; j < current_panel_cols; ++j) {
            // h_panel_ipiv[j] is 1-based relative to panel start
            // Convert to global 1-based index: panel_start + panel_relative_index
            h_IPIV[k + j] = h_panel_ipiv[j] + k;
        }
        delete[] h_panel_ipiv;

        // b.iii. Copy updated panel back for FP64 factorization
        hipMemcpy2D(d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            d_A + k * N + k, N * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);



            
        // Panel LU in FP64 (no pivoting, kernel)
        if (threads > 0) {
            dgetf2_native_npv<<<1, threads>>>(panel_rows, current_panel_cols, d_P_FP64_NPV_buffer, panel_rows);
            hipDeviceSynchronize();
        }

        // Copy back the panel to d_A
        hipMemcpy2D(d_A + k * N + k, N * sizeof(double),
            d_P_FP64_NPV_buffer, panel_rows * sizeof(double),
            current_panel_cols * sizeof(double), panel_rows,
            hipMemcpyDeviceToDevice);

        // c. Trailing submatrix update (cuBLAS)
        if (k + current_panel_cols < N) {
            int m = panel_rows - current_panel_cols;
            int n = N - k - current_panel_cols;
            DTRSM_cublas(
                handle,
                d_A + k * N + k + current_panel_cols,
                N,
                d_A + (k + current_panel_cols) * N + k + current_panel_cols,
                N,
                m,
                current_panel_cols
            );
            DGEMM_cublas(
                handle,
                d_A + (k + current_panel_cols) * N + k + current_panel_cols,
                N,
                d_A + (k + current_panel_cols) * N + k,
                N,
                d_A + k * N + k + current_panel_cols,
                N,
                m,
                n,
                current_panel_cols
            );
        }
    }

    // Copy result back to host
    hipMemcpy(h_A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    memcpy(A, h_A, N * N * sizeof(double));

    // Copy IPIV back to host
    memcpy(IPIV, h_IPIV, N * sizeof(int));
    // Cleanup
    // hipblasDestroy(handle);
    delete[] h_A;
    delete[] h_IPIV;
    hipFree(d_A);
    hipFree(d_P_FP16_buffer);
    hipFree(d_P_FP64_NPV_buffer);
    hipFree(d_IPIV_panel);
    hipFree(d_IPIV);
}